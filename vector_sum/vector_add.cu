
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
//perform vector addition utilizing blocks and threads

__global__ void add(int *a, int *b, int *c, int n) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < n) //avoid accessing beyond end of array
                c[index] = a[index] + b[index];

}

//populate vectors with random ints
void random_ints(int* a, int N) {
    for (int i=0; i < N; i++){
        a[i] = rand() % 1000;
    }
}

#define N (2048*2048) // overall size of the data set
#define THREADS_PER_BLOCK 512 // threads per block

int main(void) {
        int *a, *b, *c;
        int *d_a, *d_b, *d_c;
        int size = N * sizeof(int);

        //alloc space for device copies of a, b, and c
        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);

        //alloc space for host copies and setup input values
        a = (int *)malloc(size); random_ints(a, N);
        b = (int *)malloc(size); random_ints(b, N);
        c = (int *)malloc(size);

        //copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        //launch add() kernel, while avoid accessing beyond the end of the array
        add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

        //clean up
        free(a); free(b); free(c);
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        return 0;
}