#include "hip/hip_runtime.h"
// Daniel Bandala @ sep 2022
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>

#define THREADS_PER_BLOCK 512

__global__ void dot_product(float* a,float* b,float* c){
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    temp[threadIdx.x] = a[index]*b[index];
    // wait to all threads to finish
    __syncthreads();

    if (threadIdx.x==0){
        int sum=0;
        for (int i=0; i<THREADS_PER_BLOCK ; i++)
            sum += temp[i];
        
        atomicAdd(c,sum);
    }
}


// Helper function for using CUDA to operate vectors in parallel.
extern "C" hipError_t dotProductCUDA(std::vector<float>& a, std::vector<float>& b, float& c, unsigned int size)
{
    float *dev_a = 0;
    float *dev_b = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Display device parameters
    int nDevices;
    hipGetDeviceCount(&nDevices);
    printf("Number of devices: %d\n", nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
            prop.memoryClockRate / 1024);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n", (float)(prop.totalGlobalMem) / 1024.0 / 1024.0 / 1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n", (float)(prop.sharedMemPerBlock) / 1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n", prop.deviceOverlap ? "yes" : "no");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a.data(), size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b.data(), size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_c, &c, sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dot_product <<<ceil(size/THREADS_PER_BLOCK), 4>>>(dev_c, dev_a, dev_b);
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dotProductKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching dotProductKernel! (%s)\n", cudaStatus,hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}